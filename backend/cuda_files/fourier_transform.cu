#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

__global__ void elementwiseMulAndScale(hipfftComplex *A, hipfftComplex *B, int size, float scale) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < size) {
        hipfftComplex a = A[i];
        hipfftComplex b = B[i];
        A[i] = make_hipFloatComplex((a.x * b.x - a.y * b.y) * scale, (a.x * b.y + a.y * b.x) * scale);
    }
}

extern "C" {

    void complexMulFFT(float *A, float *B, float *C, int size) {
        int total = size;
        int msize = total * sizeof(hipfftComplex);

        // Allocate host memory for complex numbers:
        hipfftComplex *h_A = reinterpret_cast<hipfftComplex*>(A);
        hipfftComplex *h_B = reinterpret_cast<hipfftComplex*>(B);
        hipfftComplex *h_C = reinterpret_cast<hipfftComplex*>(C);

        // Allocate device memory:
        hipfftComplex *gpu_A, *gpu_B;
        hipMalloc((void**)&gpu_A, msize);
        hipMemcpy(gpu_A, h_A, msize, hipMemcpyHostToDevice);
        hipMalloc((void**)&gpu_B, msize);
        hipMemcpy(gpu_B, h_B, msize, hipMemcpyHostToDevice);

        // Create cuFFT plan:
        hipfftHandle plan;
        hipfftPlan1d(&plan, size, HIPFFT_C2C, 1);

        // Perform FFT on both vectors:
        hipfftExecC2C(plan, gpu_A, gpu_A, HIPFFT_FORWARD);
        hipfftExecC2C(plan, gpu_B, gpu_B, HIPFFT_FORWARD);

        // Element-wise multiplication and scaling:
        int threadsPerBlock = 256;
        int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
        elementwiseMulAndScale<<<blocksPerGrid, threadsPerBlock>>>(gpu_A, gpu_B, size, 1.0f / size);

        // Inverse FFT:
        hipfftExecC2C(plan, gpu_A, gpu_A, HIPFFT_BACKWARD);

        // Copy the result back to host memory:
        hipMemcpy(h_C, gpu_A, msize, hipMemcpyDeviceToHost);

        // Cleanup:
        hipfftDestroy(plan);
        hipFree(gpu_A);
        hipFree(gpu_B);
    }
}
