#include <hip/hip_runtime.h>

__global__ void vecAdd(float *A, float *B, float *C, int N) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < N) {
        C[i] = A[i] + B[i];
    }
}

extern "C" {
    void runVecAdd(float *A, float *B, float *C, int N) {
        float *d_A, *d_B, *d_C;
        int size = N * sizeof(float);

        // Allocate device memory
        hipMalloc((void **)&d_A, size);
        hipMalloc((void **)&d_B, size);
        hipMalloc((void **)&d_C, size);

        // Copy inputs to device
        hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
        hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

        // Launch the vector addition kernel
        int threadsPerBlock = 256;
        int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
        vecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

        // Copy result back to host
        hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

        // Free device memory
        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
    }
}

